// tutorial via: https://developer.nvidia.com/blog/even-easier-introduction-cuda/
//   intro to cuda, add two elements of arrays with a million elements in each


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>



__global__
void add(int n, float* x, float* y) {
    for(int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}



int main(int argc, char* argv[]) {

    int N = 1<<20; // 1M elements (i think 1048576? 2^20)
    // std::cout << N << std::endl;

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // init arrays on host
    for(int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1m elements on cpu
    // !!NOTE!!: syntax change here, global function call with <<>>
    add<<<1, 1>>>(N, x, y);
    // first 1: number of blocks
    // second 1: number of threads per block
    // add: is the __global__ kernel function
    // <<<1, N>>> is the launch configuration

    // wait for gpu to finish
    hipDeviceSynchronize();

    // error check, array should be full of 3.0
    float maxError = 0.0f;
    for(int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "max err: " << maxError << std::endl;

    // free mem
    hipFree(x);
    hipFree(y);

    return 0;
}